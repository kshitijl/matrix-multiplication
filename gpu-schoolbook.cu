#include "hip/hip_runtime.h"
#include "cuda-common.hxx"

template<typename TT>
__global__ void mm_kernel(const TT * AA, const TT * BB, TT * CC, size_t nn) {
    unsigned int ii = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int jj = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int index = ii*nn + jj;
    if(ii < nn and jj < nn) {
        CC[index] = 0;
        for(int kk = 0; kk < nn; ++kk)
            CC[index] += AA[ii*nn + kk] * BB[kk*nn + jj];
    }
}

template<typename TT>
void matrix_multiply(const TT * AA, const TT * BB, TT * CC, size_t nn) {
    TT *da, *db, *dc;

    CUDA_CALL(hipMalloc((void **) &da, sizeof(TT)*nn*nn));
    CUDA_CALL(hipMalloc((void **) &db, sizeof(TT)*nn*nn));
    CUDA_CALL(hipMalloc((void **) &dc, sizeof(TT)*nn*nn));

    CUDA_CALL(hipMemcpy(da, AA, sizeof(TT)*nn*nn, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(db, BB, sizeof(TT)*nn*nn, hipMemcpyHostToDevice));

    dim3 dimGrid(ceil(nn/32.0), ceil(nn/32.0));
    dim3 dimBlock(32,32);
    mm_kernel<<< dimGrid, dimBlock >>>(da, db, dc, nn);
    CUDA_CALL(hipPeekAtLastError());
    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipMemcpy(CC, dc, sizeof(TT)*nn*nn, hipMemcpyDeviceToHost));
    
    CUDA_CALL(hipFree(da));
    CUDA_CALL(hipFree(db));
    CUDA_CALL(hipFree(dc));
}

template void matrix_multiply<float>(const float*, const float*, float*, size_t);
template void matrix_multiply<double>(const double*, const double*, double*, size_t);
